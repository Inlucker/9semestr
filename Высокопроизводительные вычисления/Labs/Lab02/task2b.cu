#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <iostream>
#include <string>

#define N (500)
#define THREADS_N (32)
#define ITERS (100)

void randMtrx(float* mtrx, int n)
{
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      mtrx[n * j + i] = rand() % 10;
}

float* createMtrx(int n)
{
  float* mtrx = (float*)malloc(N * N * sizeof(float));
  randMtrx(mtrx, n);
  return mtrx;
}

float* createMtrxUnified(int n)
{
  float* mtrx;
  // Unified Memory
  hipMallocManaged((void**)&mtrx, N * N * sizeof(float));
  randMtrx(mtrx, n);
  return mtrx;
}

float* createMtrxOnDevice(int n)
{
  float* mtrx_dev = NULL;
  hipMalloc((void**)&mtrx_dev, n * n * sizeof(float));
  return mtrx_dev;
}

/*float* copyMtrxToDevice(int n, float*& mtrx)
{
  float* mtrx_dev = NULL;
  hipMalloc((void**)&mtrx_dev, n * n * sizeof(float));
  hipMemcpy(mtrx_dev, mtrx, n * n * sizeof(float), hipMemcpyHostToDevice);
  return mtrx_dev;
}*/

float* copyMtrxFromDevice(int n, float*& mtrx_dev)
{
  float* mtrx = (float*)malloc(N * N * sizeof(float));
  hipMemcpy(mtrx, mtrx_dev, N * N * sizeof(float), hipMemcpyDeviceToHost);
  return mtrx;
}

void deleteMtrx(float*& mtrx)
{
  free(mtrx);
  mtrx = NULL;
}

void deleteMtrxFromDevice(float*& mtrx_dev)
{
  if (hipFree(mtrx_dev) != hipSuccess)
    printf("Error in deleteMtrxFromDevice()");
  mtrx_dev = NULL;
}

void printMtrx(float* mtrx, int n)
{
  for (int i = 0; i < n; i++)
  {
    for (int j = 0; j < n; j++)
      printf("%3.0f ", mtrx[n * j + i]);
    printf("\n");
  }
  printf("\n");
}

long double getGflops(long long n, double time)
{
  long double fl_opers = 1e-9;
  fl_opers *= n * n * n * 2;
  return fl_opers / time;
}

void seq_dgemm(int n, float* a, float* b, float* c)
{
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
    {
      c[n * j + i] = 0;
      for (int k = 0; k < n; k++)
        c[n * j + i] += (a[n * k + i] * b[n * j + k]);
    }
}

__global__ void cuda_dgemm(int n, float* a, float* b, float* c)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  //printf("%d, %d\n", idx, idy);
  if (idx >= N || idy >= N)
    return;
  int i = idx;
  int j = idy;

  float res = 0;
  for (int k = 0; k < n; k++)
    res += (a[n * k + i] * b[n * j + k]);
  c[n * j + i] = res;
}

bool compareMtrx(int n, float* a, float* b)
{
  for (int i = 0; i < n * n; i++)
    if (a[i] != b[i])
    {
      int i2 = i / N;
      int j = i % N;
      printf("compareMtrx() Error: i = %d, j = %d, %.0f != %.0f\n", i2, j, a[i], b[i]);
      return false;
    }
  return true;
}

__global__ void cuda_hello()
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  printf("Hello World from GPU! %d %d\n", idx, idy);
}

int main()
{
  srand(time(NULL));
  printf("N = %d\n", N);
  dim3 threads(THREADS_N, THREADS_N);
  printf("threads.x = %d threads.y = %d\n", threads.x, threads.y);
  dim3 blocks((N - 1) / THREADS_N + 1, (N - 1) / THREADS_N + 1);
  printf("blocks.x = %d blocks.y = %d\n\n", blocks.x, blocks.y);

  float* a_unif = createMtrxUnified(N);
  float* b_unif = createMtrxUnified(N);
  float* c_unif = createMtrxUnified(N);
  float* c = copyMtrxFromDevice(N, c_unif);

  //printMtrx(a, N);
  //printMtrx(b, N);
  seq_dgemm(N, a_unif, b_unif, c);
  //printMtrx(c, N);

  cuda_dgemm << < blocks, threads >> > (N, a_unif, b_unif, c_unif);
  float* d = copyMtrxFromDevice(N, c_unif);
  //printMtrx(cdev, N);

  if (compareMtrx(N, c, d))
    printf("Unified Memory cuda_dgemm() == seq_dgemm()\n\n");
  else
    printf("Unified Memory cuda_dgemm() != seq_dgemm()\n\n");

  deleteMtrx(c);
  deleteMtrx(d);
  deleteMtrxFromDevice(a_unif);
  deleteMtrxFromDevice(b_unif);
  deleteMtrxFromDevice(c_unif);

  return 0;
}