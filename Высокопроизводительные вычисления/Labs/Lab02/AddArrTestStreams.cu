#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <iostream>
#include <string>

#define N 67108864
#define THREADS_NN 256
#define CUDA_STREAMS_NUM 8
#define ITERS 10

// CUDA ���� ��� �������� ���� ��������
__global__ void addArrays(int* a, int* b, int* c, int size)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  //printf("tid = %d\n", tid);
  if (tid < size)
  {
    c[tid] = a[tid] + b[tid];
    //printf("%d = %d + %d\n", c[tid], a[tid], b[tid]);
  }
}

int div_up(int x, int y)
{
  return (x - 1) / y + 1;
}

int main()
{
  //without cuda streams
  {
    // ������� ������� ��� ��������� ������� ����������
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ������ �������
    int size = N;

    // �������� ������ �� �����
    int* h_a = (int*)malloc(size * sizeof(int));
    int* h_b = (int*)malloc(size * sizeof(int));
    int* h_c = (int*)malloc(size * sizeof(int));

    // ������������� ��������
    for (int i = 0; i < size; i++)
    {
      h_a[i] = i;
      h_b[i] = 2 * i;
    }

    // �������� ������ �� ����������
    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // ��������� ������
    hipEventRecord(start);

    // ������ ������������ ������ � �����
    int threadsPerBlock = THREADS_NN;
    //int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGrid = div_up(size, threadsPerBlock);
    //printf("threadsPerBlock = %d\nblocksPerGrid = %d\n", threadsPerBlock, blocksPerGrid);

    for (int it = 0; it < ITERS; it++)
    {
      // �������� ������ � ����� �� ����������
      hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

      // ��������� ������
      //hipEventRecord(start);

      // �������� ���� ��� �������� �������� �� ����������
      addArrays << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c, size);

      // ������������� ������
      //hipEventRecord(stop);
      //hipEventSynchronize(stop);
      // �������� ����� ����������
      //float milliseconds = 0;
      //hipEventElapsedTime(&milliseconds, start, stop);
      //printf("             addArrays() time = %f ms\n", milliseconds);

      // �������� ��������� � ���������� �� ����
      hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    // ������������� ������
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // �������� ����� ����������
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("             addArrays() time = %f ms\n", milliseconds / ITERS);

    //addArrays << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c, size);
    //hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // ������� ���������
    for (int i = 0; i < size; i++)
      if (h_a[i] + h_b[i] != h_c[i])
        printf("Error: %d + %d != %d\n", h_a[i], h_b[i], h_c[i]);

    //for (int i = 0; i < size; i++)
    //  printf("%d ", h_c[i]);
    //printf("\n");

    // ����������� ������
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  //with cuda streams
  {
    // ������� ������� ��� ��������� ������� ����������
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ������ �������
    int size = N;

    // �������� ������ �� �����
    int* h_a = (int*)malloc(size * sizeof(int));
    int* h_b = (int*)malloc(size * sizeof(int));
    int* h_c = (int*)malloc(size * sizeof(int));

    // ������������� ��������
    for (int i = 0; i < size; i++)
    {
      h_a[i] = i;
      h_b[i] = 2 * i;
    }

    // �������� ������ �� ����������
    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // �������� �������
    hipStream_t stream[CUDA_STREAMS_NUM];
    for (int i = 0; i < CUDA_STREAMS_NUM; i++)
      hipStreamCreate(&stream[i]);
    int full_mem_size = N * sizeof(int);
    int part_mem_size = full_mem_size / CUDA_STREAMS_NUM;
    int part_size = size / CUDA_STREAMS_NUM;

    // ��������� ������
    hipEventRecord(start);

    // ������ ������������ ������ � �����
    int threadsPerBlock = THREADS_NN;
    //int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    //int blocksPerGrid = div_up(size, threadsPerBlock);
    int blocksPerGrid = div_up(div_up(size, threadsPerBlock), CUDA_STREAMS_NUM);
    //printf("threadsPerBlock = %d\nblocksPerGrid = %d\n", threadsPerBlock, blocksPerGrid);

    for (int it = 0; it < ITERS; it++)
    {
      // �������� ������ � ����� �� ����������
      for (int i = 0; i < CUDA_STREAMS_NUM; i++)
      {
        hipMemcpyAsync(d_a + i * part_size, h_a + i * part_size, part_mem_size, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(d_b + i * part_size, h_b + i * part_size, part_mem_size, hipMemcpyHostToDevice, stream[i]);
      }

      // ��������� ������
      //hipEventRecord(start);

      // �������� ���� ��� �������� �������� �� ����������
      for (int i = 0; i < CUDA_STREAMS_NUM; i++)
        addArrays << <blocksPerGrid, threadsPerBlock, 0, stream[i] >> > (d_a + i * part_size, d_b + i * part_size, d_c + i * part_size, part_size);

      // ������������� ������
      //hipEventRecord(stop);
      //hipEventSynchronize(stop);
      // �������� ����� ����������
      //float milliseconds = 0;
      //hipEventElapsedTime(&milliseconds, start, stop);
      //printf("Cuda Streams addArrays() time = %f ms\n", milliseconds);

      // �������� ��������� � ���������� �� ����
      for (int i = 0; i < CUDA_STREAMS_NUM; i++)
        hipMemcpyAsync(h_c + i * part_size, d_c + i * part_size, part_mem_size, hipMemcpyDeviceToHost, stream[i]);

      if (hipDeviceSynchronize() != hipSuccess)
        printf("hipDeviceSynchronize() Error\n");
    }

    // ������������� ������
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // �������� ����� ����������
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Cuda Streams addArrays() time = %f ms\n", milliseconds / ITERS);

    //for (int i = 0; i < CUDA_STREAMS_NUM; i++)
    //  addArrays << <blocksPerGrid, threadsPerBlock, 0, stream[i] >> > (d_a + i * part_size, d_b + i * part_size, d_c + i * part_size, part_size);
    //for (int i = 0; i < CUDA_STREAMS_NUM; i++)
    //  hipMemcpyAsync(h_c + i * part_size, d_c + i * part_size, part_mem_size, hipMemcpyDeviceToHost, stream[i]);

    // ������� ���������
    for (int i = 0; i < size; i++)
      if (h_a[i] + h_b[i] != h_c[i])
        printf("Error: %d + %d != %d\n", h_a[i], h_b[i], h_c[i]);

    //for (int i = 0; i < size; i++)
    //  printf("%d ", h_c[i]);
    //printf("\n");

    // ����������� ������
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    for (int i = 0; i < CUDA_STREAMS_NUM; i++)
      hipStreamDestroy(stream[i]);
  }

  return 0;
}
