#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>

//#define N (1024)
#define N (4)
#define THREADS_PER_BLOCK 512
//#define BLOCK_SIZE 16 // submatrix size
#define BLOCK_SIZE 4 // submatrix size

void randMtrx(float* mtrx, int n)
{
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      mtrx[n * j + i] = rand() % 10;
}

float* createMtrxCuda(int n, float*& mtrx)
{
  mtrx = new float[n * n];
  randMtrx(mtrx, n);
  float* mtrx_dev = NULL;
  hipMalloc((void**)&mtrx_dev, n*n*sizeof(float));
  hipMemcpy(mtrx_dev, mtrx, n * n * sizeof(float), hipMemcpyHostToDevice);
  return mtrx_dev;
}

void deleteMtrx(float*& mtrx)
{
  delete[] mtrx;
  mtrx = NULL;
}

void deleteMtrxCuda(float*& mtrx)
{
  hipFree(mtrx);
  mtrx = NULL;
}

void deleteMtrxs(float*& mtrx, float*& mtrx_dev)
{
  deleteMtrx(mtrx);
  deleteMtrxCuda(mtrx_dev);
}

void printMtrx(float* mtrx, int n)
{
  for (int i = 0; i < n; i++)
  {
    for (int j = 0; j < n; j++)
      printf("%f ", mtrx[n * j + i]);
    printf("\n");
  }
  printf("\n");
}

long double getGflops(long long n, double time)
{
  long double fl_opers = 1e-9;
  fl_opers *= n * n * n * 2;
  return fl_opers / time;
}

void seq_dgemm(int n, float* a, float* b, float* c)
{
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
    {
      c[n * j + i] = 0;
      for (int k = 0; k < n; k++)
        c[n * j + i] += (a[n * k + i] * b[n * j + k]);
    }
}

__global__ void cuda_dgemm(int n, float* a, float* b, float* c)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int j = idy;
  int i = idx;
  c[n * j + i] = 0;
  for (int k = 0; k < n; k++)
    c[n * j + i] += (a[n * k + i] * b[n * j + k]);
}

bool compareMtrx(int n, float* a, float* b)
{
  for (int i = 0; i < n * n; i++)
    if (a[i] != b[i])
      return false;
  return true;
}

__global__ void cuda_hello()
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  printf("Hello World from GPU! %d %d\n", idx, idy);
}

int main()
{
  srand(time(NULL));
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 blocks(N / threads.x, N / threads.y);
  //cuda_hello << < threads, blocks >> > ();

  float *a = NULL, *b = NULL, *c = NULL;
  float* adev = createMtrxCuda(N, a);
  float* bdev = createMtrxCuda(N, b);
  float* cdev = createMtrxCuda(N, c);

  printMtrx(a, N);
  printMtrx(b, N);
  seq_dgemm(N, a, b, c);
  printMtrx(c, N);

  cuda_dgemm << < threads, blocks >> > (N, adev, bdev, cdev);
  float* d = new float[N * N];
  hipMemcpy(d, cdev, N*N*sizeof(float), hipMemcpyDeviceToHost);
  printMtrx(d, N);

  if (compareMtrx(N, c, d))
    printf("cuda_dgemm() == seq_dgemm()\n");
  else
    printf("cuda_dgemm() != seq_dgemm()\n");

  deleteMtrxs(a, adev);
  deleteMtrxs(b, bdev);
  deleteMtrxs(c, cdev);
  delete[] d;

  //cuda_hello << < N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > ();
  //printf("Hello World from CPU!\n");
  return 0;
}

/*
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/